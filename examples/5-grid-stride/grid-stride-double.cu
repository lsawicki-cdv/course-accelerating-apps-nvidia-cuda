
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
    {
        a[i] = i;
    }
}

/*
 * In the current application, `N` is larger than the grid.
 * Refactor this kernel to use a grid-stride loop in order that
 * each parallel thread work on more than one element of the array.
 */

__global__ void doubleElements(int *a, int N)
{
    // NO GRID STRIDE CODE
    // int i;
    // i = blockIdx.x * blockDim.x + threadIdx.x;
    // if (i < N)
    // {
    //     a[i] *= 2;
    // }

    /*
     * Use a grid-stride loop so each thread does work
     * on more than one element in the array.
     */

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < N; i += stride)
    {
        a[i] *= 2;
    }
}

bool checkElementsAreDoubled(int *a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
    {
        if (a[i] != i * 2)
            return false;
    }
    return true;
}

int main()
{
    /*
     * `N` is greater than the size of the grid.
     */

    int N = 10000;
    int *a;

    size_t size = N * sizeof(int);
    hipMallocManaged(&a, size);

    init(a, N);

    /*
     * The size of this grid is 256*32 = 8192.
     */

    size_t threads_per_block = 256;
    size_t number_of_blocks = 32;

    doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
    hipDeviceSynchronize();

    bool areDoubled = checkElementsAreDoubled(a, N);
    printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

    hipFree(a);
}
