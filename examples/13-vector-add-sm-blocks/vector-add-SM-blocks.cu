
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addArraysInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for (int i = 0; i < N; i++)
  {
    if (array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  const int N = 2 << 24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;

  numberOfBlocks = (N + numberOfBlocks - 1) / numberOfBlocks;

  hipError_t addArraysErr;
  hipError_t asyncErr;

  addArraysInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addArraysErr = hipGetLastError();
  if (addArraysErr != hipSuccess)
  {
    printf("Error: %s\n", hipGetErrorString(addArraysErr));
  }

  asyncErr = hipDeviceSynchronize();
  if (asyncErr != hipSuccess)
  {
    printf("Error: %s\n", hipGetErrorString(asyncErr));
  }

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
