
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
    {
        a[i] = i;
    }
}

__global__ void doubleElements(int *a, int N)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Change the for loop to catch an error
    // for (int i = idx; i < N + stride * 32; i += stride)
    for (int i = idx; i < N; i += stride)
    {
        a[i] *= 2;
    }
}

bool checkElementsAreDoubled(int *a, int N)
{
    int i;
    for (i = 0; i < N; ++i)
    {
        if (a[i] != i * 2)
            return false;
    }
    return true;
}

int main()
{
    int N = 10000;
    int *a;

    size_t size = N * sizeof(int);
    hipMallocManaged(&a, size);

    init(a, N);

    // Change the number of threads per block to trigger error
    size_t threads_per_block = 1024;
    // size_t threads_per_block = 2048;

    size_t number_of_blocks = 32;

    hipError_t syncErr, asyncErr;

    doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);

    /*
     * Catch errors for both the kernel launch above and any
     * errors that occur during the asynchronous `doubleElements`
     * kernel execution.
     */

    syncErr = hipGetLastError();
    asyncErr = hipDeviceSynchronize();

    if (syncErr != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(syncErr));
    }

    if (asyncErr != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(asyncErr));
    }

    bool areDoubled = checkElementsAreDoubled(a, N);
    printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

    hipFree(a);
}
