
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printNumber(int number)
{
  printf("%d\n", number);
}

int main()
{
  for (int i = 0; i < 5; ++i)
  {
    hipStream_t stream;
    hipStreamCreate(&stream);
    printNumber<<<2, 2, 0, stream>>>(i);
    hipStreamDestroy(stream);
  }
  hipDeviceSynchronize();
}
