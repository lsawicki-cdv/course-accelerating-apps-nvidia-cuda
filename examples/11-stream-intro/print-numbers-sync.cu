
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printNumber(int number)
{
  printf("%d\n", number);
}

int main()
{
  for (int i = 0; i < 5; ++i)
  {
    printNumber<<<2, 2>>>(i);
  }
  hipDeviceSynchronize();
}
