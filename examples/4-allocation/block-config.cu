
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N)
    {
        a[i] = initialValue;
    }
}

int main()
{
    int N = 1000;

    int *a;
    size_t size = N * sizeof(int);

    hipMallocManaged(&a, size);

    size_t threads_per_block = 256;

    /*
     * The following is idiomatic CUDA to make sure there are at
     * least as many threads in the grid as there are `N` elements.
     */

    size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

    printf("Number of blocks: %ld \n", number_of_blocks);

    int initialValue = 6;

    initializeElementsTo<<<number_of_blocks, threads_per_block>>>(initialValue, a, N);
    hipDeviceSynchronize();

    /*
     * Check to make sure all values in `a`, were initialized.
     */

    for (int i = 0; i < N; ++i)
    {
        if (a[i] != initialValue)
        {
            printf("FAILURE: target value: %d\t a[%d]: %d\n", initialValue, i, a[i]);
            hipFree(a);
            exit(1);
        }
    }
    printf("SUCCESS!\n");

    hipFree(a);
}
